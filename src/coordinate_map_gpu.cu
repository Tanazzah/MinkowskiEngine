#include "hip/hip_runtime.h"
/* Copyright (c) 2020 NVIDIA CORPORATION.
 * Copyright (c) 2018-2020 Chris Choy (chrischoy@ai.stanford.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "coordinate_map_functors.cuh"
#include "coordinate_map_gpu.cuh"
#include "gpu.cuh"
#include "kernel_map.cuh"

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

namespace minkowski {

/*
 * @brief Given a key iterator begin-end pair and a value iterator begin-end
 * pair, insert all elements.
 *
 * @note The key and value iterators can be 1) pointers, 2) coordinate or vector
 * iterators.
 *
 * @return none
 */
template <typename coordinate_type, typename MapAllocator,
          typename CoordinateAllocator, typename KernelMapAllocator>
template <typename mapped_iterator>
void CoordinateMapGPU<
    coordinate_type, MapAllocator, CoordinateAllocator,
    KernelMapAllocator>::insert(coordinate_iterator<coordinate_type> key_first,
                                coordinate_iterator<coordinate_type> key_last,
                                mapped_iterator value_first,
                                mapped_iterator value_last) {
  size_type const N = key_last - key_first;
  LOG_DEBUG("key iterator length", N);
  ASSERT(N == value_last - value_first,
         "The number of items mismatch. # of keys:", N,
         ", # of values:", value_last - value_first);

  // Copy the coordinates to m_coordinate
  base_type::reserve(N);
  CUDA_CHECK(
      hipMemcpy(coordinate_data(), // dst
                 key_first->data(), // first element of the dereferenced iter.
                 sizeof(coordinate_type) * N * m_coordinate_size, // bytes
                 hipMemcpyDeviceToDevice));
  CUDA_CHECK(hipStreamSynchronize(0));
  LOG_DEBUG("Reserved and copied", N, "x", m_coordinate_size, "coordinates");

  // Insert coordinates
  thrust::counting_iterator<uint32_t> count{0};
  auto insert = detail::insert_coordinate<coordinate_type, map_type,
                                          thrust::counting_iterator<uint32_t>>{
      *m_map,                  // map
      const_coordinate_data(), // coordinates,
      value_first,             // iter begin
      m_coordinate_size};

  thrust::device_vector<bool> success(N);
  m_valid_index.resize(N);
  thrust::sequence(thrust::device, m_valid_index.begin(), m_valid_index.end());

  // Insert coordinates
  thrust::transform(count, count + N, success.begin(), insert);

  // Valid row index
  auto valid_begin = thrust::make_zip_iterator(
      thrust::make_tuple(success.begin(), m_valid_index.begin()));
  size_type const number_of_valid =
      thrust::remove_if(thrust::device, valid_begin,
                        thrust::make_zip_iterator(thrust::make_tuple(
                            success.end(), m_valid_index.end())),
                        detail::is_first<false>()) -
      valid_begin;
  m_valid_index.resize(number_of_valid);
}

/*
 * @brief given a key iterator begin-end pair find all valid keys and its
 * index.
 *
 * @return a pair of (valid index, query value) vectors.
 */
template <typename coordinate_type, typename MapAllocator,
          typename CoordinateAllocator, typename KernelMapAllocator>
std::pair<thrust::device_vector<uint32_t>, thrust::device_vector<uint32_t>>
CoordinateMapGPU<coordinate_type, MapAllocator, CoordinateAllocator,
                 KernelMapAllocator>::find(coordinate_iterator<coordinate_type>
                                               key_first,
                                           coordinate_iterator<coordinate_type>
                                               key_last) const {
  size_type N = key_last - key_first;

  LOG_DEBUG(N, "queries for find.");
  auto const find_functor = detail::find_coordinate<coordinate_type, map_type>(
      *m_map, key_first->data(), m_unused_element, m_coordinate_size);
  LOG_DEBUG("Find functor initialized.");
  auto const invalid_functor =
      detail::is_unused_pair<coordinate_type, mapped_type>(m_unused_element);
  LOG_DEBUG("Valid functor initialized.");

  thrust::counting_iterator<index_type> index{0};
  device_index_vector_type input_index(N);
  device_index_vector_type results(N);
  LOG_DEBUG("Initialized functors.");
  thrust::sequence(thrust::device, input_index.begin(), input_index.end());
  thrust::transform(thrust::device, index, index + N, results.begin(),
                    find_functor);

  size_type const number_of_valid =
      thrust::remove_if(thrust::device,
                        thrust::make_zip_iterator(thrust::make_tuple(
                            input_index.begin(), results.begin())),
                        thrust::make_zip_iterator(thrust::make_tuple(
                            input_index.end(), results.end())),
                        invalid_functor) -
      thrust::make_zip_iterator(
          thrust::make_tuple(input_index.begin(), results.begin()));
  LOG_DEBUG("Number of valid", number_of_valid);
  input_index.resize(number_of_valid);
  results.resize(number_of_valid);

  return std::make_pair(input_index, results);
}

/*
 * @brief given a key iterator begin-end pair find all valid keys and its
 * index.
 *
 * @return a pair of (valid index, query value) vectors.
 */
template <typename coordinate_type, typename MapAllocator,
          typename CoordinateAllocator, typename KernelMapAllocator>
CoordinateMapGPU<coordinate_type, MapAllocator, CoordinateAllocator,
                 KernelMapAllocator>
CoordinateMapGPU<coordinate_type, MapAllocator, CoordinateAllocator,
                 KernelMapAllocator>::stride(stride_type const &stride) const {

  // Over estimate the reserve size to be size();
  size_type const N = size();

  self_type stride_map(
      N, m_coordinate_size,
      detail::stride_tensor_stride(base_type::m_tensor_stride, stride),
      base_type::m_allocator);

  // stride coordinates
  thrust::counting_iterator<uint32_t> count_begin{0};
  thrust::for_each(
      count_begin, count_begin + N,
      detail::stride_copy<coordinate_type, index_type>(
          m_coordinate_size,
          thrust::raw_pointer_cast(stride_map.m_device_tensor_stride.data()),
          thrust::raw_pointer_cast(m_valid_index.data()),
          const_coordinate_data(), //
          stride_map.coordinate_data()));

  thrust::device_vector<bool> success(N);
  auto &stride_valid_index = stride_map.m_valid_index;
  stride_valid_index.resize(N);
  thrust::sequence(thrust::device, stride_valid_index.begin(),
                   stride_valid_index.end());

  // Insert coordinates
  auto insert = detail::insert_coordinate<coordinate_type, map_type,
                                          index_type *>{
      *stride_map.m_map,                                   // map
      stride_map.const_coordinate_data(),                  // coordinates,
      thrust::raw_pointer_cast(stride_valid_index.data()), // iter begin
      m_coordinate_size};
  thrust::transform(count_begin, count_begin + N, success.begin(), insert);

  // Valid row index
  auto valid_begin = thrust::make_zip_iterator(
      thrust::make_tuple(success.begin(), stride_valid_index.begin()));
  size_type const number_of_valid =
      thrust::remove_if(thrust::device, valid_begin,
                        thrust::make_zip_iterator(thrust::make_tuple(
                            success.end(), stride_valid_index.end())),
                        detail::is_first<false>()) -
      valid_begin;
  stride_valid_index.resize(number_of_valid);

  thrust::for_each(count_begin, count_begin + number_of_valid,
                   detail::update_value<coordinate_type, map_type>{
                       *stride_map.m_map, stride_map.const_coordinate_data(),
                       thrust::raw_pointer_cast(stride_valid_index.data()),
                       m_coordinate_size});

  return stride_map;
}

namespace detail {

template <typename coordinate_type, //
          typename size_type,       //
          typename index_type,      //
          typename map_type>
__global__ void count_kernel(map_type const __restrict__ in_map,        //
                             map_type const __restrict__ out_map,       //
                             size_type const num_map_values_per_thread, //
                             size_type const num_threads,               //
                             gpu_kernel_region<coordinate_type> kernel, //
                             index_type *__restrict__ p_count_per_thread) {
  extern __shared__ coordinate_type sh_all[];

  auto const tx = threadIdx.x;
  auto const bx = blockIdx.x;
  auto const x = blockDim.x * bx + tx;

  size_type coordinate_size = kernel.coordinate_size();
  size_type volume = kernel.volume();

  // clang-format off
  size_type *sh_size = reinterpret_cast<size_type *>(sh_all);

  size_type *sh_tensor_stride = sh_size;
  size_type *sh_kernel_size   = sh_tensor_stride + coordinate_size;
  size_type *sh_dilation      = sh_kernel_size   + coordinate_size;

  coordinate_type *sh_coordinate = reinterpret_cast<coordinate_type *>(sh_dilation + coordinate_size);
  coordinate_type *sh_tmp = sh_coordinate +                   tx  * coordinate_size;
  coordinate_type *sh_lb  = sh_coordinate + (1 * blockDim.x + tx) * coordinate_size;
  coordinate_type *sh_ub  = sh_coordinate + (2 * blockDim.x + tx) * coordinate_size;
  // clang-format on

  auto const equal = out_map.get_key_equal();

  // kernel_maps
  for (index_type i = tx; i < coordinate_size - 1; i += blockDim.x) {
    sh_tensor_stride[i] = kernel.tensor_stride()[i];
    sh_kernel_size[i] = kernel.kernel_size()[i];
    sh_dilation[i] = kernel.dilation()[i];
  }

  __syncthreads();

  // clang-format off
  auto const unused_key = out_map.get_unused_key();
  auto const max_index = umin((x + 1) * num_map_values_per_thread, out_map.capacity());
  // iterate over values
  size_type count = 0;
  for (index_type value_index = x * num_map_values_per_thread;
       value_index < max_index;
       ++value_index) {
    // clang-format on
    typename map_type::value_type const &out_value =
        out_map.data()[value_index];
    if (!equal(out_value.first, unused_key)) {
      // set bounds for the valid keys
      kernel.set_bounds(out_value.first.data(), sh_lb, sh_ub, sh_tmp);
      for (auto const &coordinate : kernel) {
        if (in_map.find(coordinate) != in_map.end()) {
          ++count;
        }
      }
    }
  }

  if (x < num_threads)
    p_count_per_thread[x] = count;
}

template <typename coordinate_type, //
          typename size_type,       //
          typename index_type,      //
          typename map_type>
__global__ void preallocated_kernel_map_iteration(
    map_type const __restrict__ in_map,                  //
    map_type const __restrict__ out_map,                 //
    size_type const num_map_values_per_thread,           //
    gpu_kernel_region<coordinate_type> kernel,           //
    index_type const *inclusive_count_cumsum_per_thread, //
    index_type *__restrict__ p_kernels, index_type *__restrict__ p_in_maps,
    index_type *__restrict__ p_out_maps) {
  extern __shared__ coordinate_type sh_all[];

  auto const tx = threadIdx.x;
  auto const bx = blockIdx.x;
  auto const x = blockDim.x * bx + tx;

  size_type coordinate_size = kernel.coordinate_size();
  size_type volume = kernel.volume();

  // clang-format off
  size_type *sh_size = reinterpret_cast<size_type *>(sh_all);

  size_type *sh_tensor_stride = sh_size;
  size_type *sh_kernel_size   = sh_tensor_stride + coordinate_size;
  size_type *sh_dilation      = sh_kernel_size   + coordinate_size;

  coordinate_type *sh_coordinate = reinterpret_cast<coordinate_type *>(sh_dilation + coordinate_size);
  coordinate_type *sh_tmp = sh_coordinate +                   tx  * coordinate_size;
  coordinate_type *sh_lb  = sh_coordinate + (1 * blockDim.x + tx) * coordinate_size;
  coordinate_type *sh_ub  = sh_coordinate + (2 * blockDim.x + tx) * coordinate_size;
  // clang-format on

  auto const equal = out_map.get_key_equal();

  for (index_type i = tx; i < coordinate_size - 1; i += blockDim.x) {
    sh_tensor_stride[i] = kernel.tensor_stride()[i];
    sh_kernel_size[i] = kernel.kernel_size()[i];
    sh_dilation[i] = kernel.dilation()[i];
  }

  __syncthreads();

  // clang-format off
  auto const unused_key = out_map.get_unused_key();
  auto const max_index = umin((x + 1) * num_map_values_per_thread, out_map.capacity());

  // iterate over values
  auto kernel_map_index = (x < 1) ? 0 : inclusive_count_cumsum_per_thread[x - 1];
  index_type kernel_index = 0;
  for (index_type value_index = x * num_map_values_per_thread;
       value_index < max_index;
       ++value_index) {
    typename map_type::value_type const &out_value = out_map.data()[value_index];
    // clang-format on
    if (!equal(out_value.first, unused_key)) {
      // set bounds for the valid keys
      kernel.set_bounds(out_value.first.data(), sh_lb, sh_ub, sh_tmp);
      kernel_index = 0;
      for (auto const &coordinate : kernel) {
        auto const &in_result = in_map.find(coordinate);
        if (in_result != in_map.end()) {
          // insert to
          p_kernels[kernel_map_index] = kernel_index;
          p_in_maps[kernel_map_index] = (*in_result).second;
          p_out_maps[kernel_map_index] = out_value.second;
          ++kernel_map_index;
        }
        ++kernel_index;
      }
    }
  }
}

} // namespace detail

template <typename coordinate_type, typename MapAllocator,
          typename CoordinateAllocator, typename KernelMapAllocator>
CoordinateMapGPU<coordinate_type, MapAllocator, CoordinateAllocator,
                 KernelMapAllocator>::kernel_map_type
CoordinateMapGPU<coordinate_type, MapAllocator, CoordinateAllocator,
                 KernelMapAllocator>::
    kernel_map(self_type const &out_coordinate_map,
               gpu_kernel_region<coordinate_type> const &kernel,
               uint32_t num_map_values_per_thread, uint32_t thread_dim) const {
  // Over estimate the reserve size to be size();
  size_type out_capacity = out_coordinate_map.m_map->capacity();
  size_type kernel_volume = kernel.volume();

  // clang-format off
  // (THREAD * 3 * D +  3 * D + 2 * K) * 4
  uint32_t shared_memory_size_in_bytes =
      3 * m_coordinate_size * sizeof(index_type) + // stride, kernel, dilation
      3 * thread_dim * m_coordinate_size * sizeof(coordinate_type); // tmp, lb, ub
  // clang-format on
  auto const block_dim = GET_BLOCKS(
      out_coordinate_map.capacity() / num_map_values_per_thread, thread_dim);
  size_type const num_threads =
      (out_coordinate_map.capacity() + num_map_values_per_thread - 1) /
      num_map_values_per_thread;

  LOG_DEBUG("block dim", block_dim);
  LOG_DEBUG("out_coordinate_map capacity", out_coordinate_map.capacity());
  LOG_DEBUG("shared_memory size", shared_memory_size_in_bytes);
  LOG_DEBUG("threads dim", thread_dim);
  LOG_DEBUG("num threads", num_threads);

  index_type *d_p_count_per_thread =
      m_kernel_map_allocator.allocate(num_threads);

  // clang-format off
  // Initialize count per thread
  detail::count_kernel<coordinate_type, size_type, index_type, map_type>
      <<<block_dim, thread_dim, shared_memory_size_in_bytes>>>(
          *m_map,                    //
          *out_coordinate_map.m_map, //
          num_map_values_per_thread, //
          num_threads,               //
          kernel,                    //
          d_p_count_per_thread);
  // clang-format on

  LOG_DEBUG("count_kernel finished");
  thrust::inclusive_scan(thrust::device, d_p_count_per_thread,
                         d_p_count_per_thread + num_threads,
                         d_p_count_per_thread);
  index_type num_kernel_map; // type following the kernel map allocator
  CUDA_CHECK(hipMemcpy(&num_kernel_map, d_p_count_per_thread + num_threads - 1,
                        sizeof(index_type), hipMemcpyDeviceToHost));

  // set kernel map
  LOG_DEBUG("Found", num_kernel_map, "kernel map elements.");

  kernel_map_type kernel_map(num_kernel_map);
  detail::preallocated_kernel_map_iteration<coordinate_type, size_type,
                                            index_type, map_type>
      <<<block_dim, thread_dim, shared_memory_size_in_bytes>>>(
          *m_map,                     //
          *out_coordinate_map.m_map,  //
          num_map_values_per_thread,  //
          kernel,                     //
          d_p_count_per_thread,       //
          kernel_map.kernels.begin(), //
          kernel_map.in_maps.begin(), //
          kernel_map.out_maps.begin());

  CUDA_CHECK(hipStreamSynchronize(0));
  LOG_DEBUG("Preallocated kernel map done");

  kernel_map.decompose();
  m_kernel_map_allocator.deallocate(d_p_count_per_thread, num_threads);
  LOG_DEBUG("hipFree");

  return kernel_map;
}

// Template instantiation
template class CoordinateMapGPU<default_types::dcoordinate_type>;
// Insert arg templates
using citer32 = coordinate_iterator<default_types::dcoordinate_type>;
template void CoordinateMapGPU<default_types::dcoordinate_type>::insert<
    thrust::counting_iterator<default_types::index_type>>(
    citer32,                                              // key bein
    citer32,                                              // key end
    thrust::counting_iterator<default_types::index_type>, // value begin
    thrust::counting_iterator<default_types::index_type>  // value end
);

} // namespace minkowski
