#include "hip/hip_runtime.h"
/* Copyright (c) 2020 NVIDIA CORPORATION.
 * Copyright (c) 2018-2020 Chris Choy (chrischoy@ai.stanford.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "coordinate_map_cpu.hpp"
#include "coordinate_map_key.hpp"
#include "coordinate_map_manager.hpp"
#include "types.hpp"
#include "utils.hpp"

#include "allocators.cuh"
#include "coordinate_map_gpu.cuh"

#include <torch/extension.h>

namespace minkowski {

using coordinate_type = int32_t;
using index_type = default_types::index_type;
using size_type = default_types::size_type;
using stride_type = default_types::stride_type;

#ifndef CPU_ONLY
std::pair<py::object, std::pair<at::Tensor, at::Tensor>>
coordinate_map_manager_test(const torch::Tensor &coordinates,
                            std::string string_id) {
  // Create TensorArgs. These record the names and positions of each tensor as a
  // parameter.
  torch::TensorArg arg_coordinates(coordinates, "coordinates", 0);

  torch::CheckedFrom c = "coordinate_test";
  torch::checkContiguous(c, arg_coordinates);
  // must match coordinate_type
  torch::checkScalarType(c, arg_coordinates, torch::kInt);
  torch::checkBackend(c, arg_coordinates.tensor, torch::Backend::CUDA);
  torch::checkDim(c, arg_coordinates, 2);

  auto const D = (index_type)coordinates.size(1);

  CoordinateMapManager<coordinate_type, detail::c10_allocator, CoordinateMapGPU>
      manager;

  stride_type tensor_stride;
  for (index_type i = 0; i < D - 1; ++i) {
    tensor_stride.push_back(1);
  }

  return manager.insert_and_map(coordinates, tensor_stride, string_id);
}
#endif

} // namespace minkowski

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  py::enum_<minkowski::GPUMemoryAllocatorBackend::Type>(m, "GPUMemoryAllocator")
      .value("PYTORCH", minkowski::GPUMemoryAllocatorBackend::Type::PYTORCH)
      .value("CUDA", minkowski::GPUMemoryAllocatorBackend::Type::CUDA)
      .export_values();

  py::enum_<minkowski::CoordinateMapBackend::Type>(m, "CoordinateMap")
      .value("CPU", minkowski::CoordinateMapBackend::Type::CPU)
      .value("PYTORCH", minkowski::CoordinateMapBackend::Type::CUDA)
      .export_values();

  py::class_<minkowski::CoordinateMapKey>(m, "CoordinateMapKey")
      .def(py::init<minkowski::default_types::size_type>())
      .def(py::init<minkowski::default_types::stride_type, std::string>())
      .def("__repr__", &minkowski::CoordinateMapKey::to_string)
      .def("get_dimension", &minkowski::CoordinateMapKey::get_dimension)
      .def("get_key", &minkowski::CoordinateMapKey::get_key)
      .def("set_key", (void (minkowski::CoordinateMapKey::*)(
                          minkowski::default_types::stride_type, std::string)) &
                          minkowski::CoordinateMapKey::set_key)
      .def("get_tensor_stride",
           &minkowski::CoordinateMapKey::get_tensor_stride);

  py::class_<minkowski::CoordinateMapManager<
      int32_t, minkowski::detail::c10_allocator, minkowski::CoordinateMapGPU>>(
      m, "CoordinateMapManager")
      .def(py::init<>())
      .def("insert_and_map", &minkowski::CoordinateMapManager<
                                 int32_t, minkowski::detail::c10_allocator,
                                 minkowski::CoordinateMapGPU>::insert_and_map);

  m.def("coordinate_map_manager_test", &minkowski::coordinate_map_manager_test,
        "Minkowski Engine coordinate map manager test");
}
