#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 NVIDIA CORPORATION.
 * Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "coordinate_map_gpu.cuh"
#include "coordinate_map_manager.cpp"
#include "coordinate_map_manager.hpp"

#include <ATen/cuda/HIPContext.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

namespace minkowski {

namespace detail {

template <typename src_type, typename dst_type>
__global__ void cuda_copy_n(src_type const *src, uint32_t N, dst_type *dst) {
  CUDA_KERNEL_LOOP(index, N) { dst[index] = src[index]; }
}

template <typename coordinate_type,
          template <typename C> class TemplatedAllocator>
struct insert_and_map_functor<coordinate_type, TemplatedAllocator,
                              CoordinateMapGPU> {

  std::pair<at::Tensor, at::Tensor>
  operator()(coordinate_map_key_type &map_key, at::Tensor const &th_coordinate,
             CoordinateMapManager<coordinate_type, TemplatedAllocator,
                                  CoordinateMapGPU> &manager) {
    uint32_t const N = th_coordinate.size(0);
    uint32_t const coordinate_size = th_coordinate.size(1);
    coordinate_type *p_coordinate = th_coordinate.data_ptr<coordinate_type>();

    auto coordinate_map = CoordinateMapGPU<coordinate_type, TemplatedAllocator>(
        N, coordinate_size, manager.m_gpu_default_occupancy, map_key.first);

    auto input_coordinate_range =
        coordinate_range<coordinate_type>(N, coordinate_size, p_coordinate);

    auto map_inverse_map = coordinate_map.template insert_and_map<true>(
        input_coordinate_range.begin(), input_coordinate_range.end());
    LOG_DEBUG("mapping size:", map_inverse_map.first.size());

    // insert moves map
    manager.insert(map_key, coordinate_map);

    auto const &mapping = map_inverse_map.first;
    auto const &inverse_mapping = map_inverse_map.second;

    // return tensors
    // TODO int64_t
    LOG_DEBUG("Reserve mapping torch output tensors.");
    at::Tensor th_mapping = torch::empty(
        {(int64_t)mapping.size()},
        th_coordinate.options().requires_grad(false).dtype(torch::kInt64));
    at::Tensor th_inverse_mapping = torch::empty(
        {(int64_t)inverse_mapping.size()},
        th_coordinate.options().requires_grad(false).dtype(torch::kInt64));

    auto const num_blocks =
        (mapping.size() + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;

    LOG_DEBUG("cuda_copy_n with num_blocks:", num_blocks,
              "mapping.size():", mapping.size());
    detail::cuda_copy_n<default_types::index_type, int64_t>
        <<<num_blocks, CUDA_NUM_THREADS>>>(
            thrust::raw_pointer_cast(mapping.data()), mapping.size(),
            th_mapping.data_ptr<int64_t>());

    auto const num_inv_blocks =
        (inverse_mapping.size() + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;

    LOG_DEBUG("cuda_copy_n with num_inv_blocks:", num_inv_blocks,
              "inverse_mapping.size():", inverse_mapping.size());
    detail::cuda_copy_n<default_types::index_type, int64_t>
        <<<num_inv_blocks, CUDA_NUM_THREADS>>>(
            thrust::raw_pointer_cast(inverse_mapping.data()),
            inverse_mapping.size(), th_inverse_mapping.data_ptr<int64_t>());
    CUDA_CHECK(hipStreamSynchronize(0));

    return std::make_pair(std::move(th_mapping), std::move(th_inverse_mapping));
  }
};

template <typename coordinate_type,
          template <typename C> class TemplatedAllocator>
struct kernel_map_functor<
    coordinate_type, TemplatedAllocator, CoordinateMapGPU,
    gpu_kernel_map<default_types::index_type, TemplatedAllocator<char>>> {

  gpu_kernel_map<default_types::index_type, TemplatedAllocator<char>>
  operator()(
      CoordinateMapGPU<coordinate_type, TemplatedAllocator> const &in_map,
      CoordinateMapGPU<coordinate_type, TemplatedAllocator> const &out_map,
      CUDAKernelMapMode::Mode kernel_map_mode,
      cpu_kernel_region<coordinate_type> &kernel) {
    LOG_DEBUG("cpu_kernel_region initialized with volume", kernel.volume());
    kernel.to_gpu();
    auto gpu_kernel = gpu_kernel_region<coordinate_type>(kernel);
    LOG_DEBUG("gpu_kernel_region initialization");

    return in_map.kernel_map(out_map, gpu_kernel, kernel_map_mode,
                             CUDA_NUM_THREADS);
  }
};

template <typename coordinate_type,
          template <typename C> class TemplatedAllocator>
struct stride_map_functor<
    coordinate_type, TemplatedAllocator, CoordinateMapGPU,
    gpu_kernel_map<default_types::index_type, TemplatedAllocator<char>>> {

  gpu_kernel_map<default_types::index_type, TemplatedAllocator<char>>
  operator()(
      CoordinateMapGPU<coordinate_type, TemplatedAllocator> const &in_map,
      CoordinateMapGPU<coordinate_type, TemplatedAllocator> const &out_map,
      default_types::stride_type const &stride) {
    return in_map.stride_map(out_map, stride, CUDA_NUM_THREADS);
  }
};

// a partial specialization functor for kernel map in/out swap
template <>
struct swap_in_out_map_functor<gpu_kernel_map<
    default_types::index_type, detail::default_allocator<char>>> {
  using gpu_kernel_map_type = gpu_kernel_map<default_types::index_type,
                                             detail::default_allocator<char>>;

  gpu_kernel_map_type operator()(gpu_kernel_map_type const &kernel_map) {
    return kernel_map.swap();
  }
};
template <>
struct swap_in_out_map_functor<
    gpu_kernel_map<default_types::index_type, detail::c10_allocator<char>>> {
  using gpu_kernel_map_type =
      gpu_kernel_map<default_types::index_type, detail::c10_allocator<char>>;

  gpu_kernel_map_type operator()(gpu_kernel_map_type const &kernel_map) {
    return kernel_map.swap();
  }
};

} // namespace detail

template class CoordinateMapManager<int32_t, detail::default_allocator,
                                    CoordinateMapGPU>;
template class CoordinateMapManager<int32_t, detail::c10_allocator,
                                    CoordinateMapGPU>;

} // end namespace minkowski
