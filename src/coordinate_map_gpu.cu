#include "hip/hip_runtime.h"
/* Copyright (c) 2020 NVIDIA CORPORATION.
 * Copyright (c) 2018-2020 Chris Choy (chrischoy@ai.stanford.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "coordinate_map_functors.cuh"
#include "coordinate_map_gpu.cuh"
#include "gpu.cuh"

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

namespace minkowski {

/*
 * @brief Given a key iterator begin-end pair and a value iterator begin-end
 * pair, insert all elements.
 *
 * @note The key and value iterators can be 1) pointers, 2) coordinate or vector
 * iterators.
 *
 * @return none
 */
template <typename coordinate_type, typename MapAllocator,
          typename CoordinateAllocator, typename IndexAllocator>
template <typename mapped_iterator>
void CoordinateMapGPU<
    coordinate_type, MapAllocator, CoordinateAllocator,
    IndexAllocator>::insert(coordinate_iterator<coordinate_type> key_first,
                            coordinate_iterator<coordinate_type> key_last,
                            mapped_iterator value_first,
                            mapped_iterator value_last) {
  size_type const N = key_last - key_first;
  LOG_DEBUG("key iterator length", N);
  ASSERT(N == value_last - value_first,
         "The number of items mismatch. # of keys:", N,
         ", # of values:", value_last - value_first);

  // Copy the coordinates to m_coordinate
  base_type::reserve(N);
  CUDA_CHECK(
      hipMemcpy(coordinate_data(), // dst
                 key_first->data(), // first element of the dereferenced iter.
                 sizeof(coordinate_type) * N * m_coordinate_size, // bytes
                 hipMemcpyDeviceToDevice));
  CUDA_CHECK(hipStreamSynchronize(0));
  LOG_DEBUG("Reserved and copied", N, "x", m_coordinate_size);

  // Insert coordinates
  thrust::counting_iterator<uint32_t> count{0};
  auto insert = detail::insert_coordinate<coordinate_type, map_type,
                                          thrust::counting_iterator<uint32_t>>{
      *m_map,            // map
      coordinate_data(), // coordinates,
      value_first,       // iter begin
      m_coordinate_size};
  thrust::for_each(count, count + N, insert);
}

/*
 * @brief given a key iterator begin-end pair find all valid keys and its
 * index.
 *
 * @return a pair of (valid index, query value) vectors.
 */
template <typename coordinate_type, typename MapAllocator,
          typename CoordinateAllocator, typename IndexAllocator>
thrust::pair<thrust::device_vector<uint32_t>, thrust::device_vector<uint32_t>>
CoordinateMapGPU<
    coordinate_type, MapAllocator, CoordinateAllocator,
    IndexAllocator>::find(coordinate_iterator<coordinate_type> key_first,
                          coordinate_iterator<coordinate_type> key_last) const {
  size_type N = key_last - key_first;

  // reserve the result slots
  index_vector_type valid_query_index, query_result;
  valid_query_index.reserve(N);
  query_result.reserve(N);

  LOG_DEBUG(N, "queries for find.")
  auto const find_functor = detail::find_coordinate<coordinate_type, map_type>(
      *m_map, key_first->data(), m_unused_element, m_coordinate_size);
  LOG_DEBUG("Find functor initialized.")
  auto const invalid_functor =
      detail::is_invalid_pair<coordinate_type, mapped_type>(m_unused_element);
  LOG_DEBUG("Valid functor initialized.")

  thrust::counting_iterator<index_type> index{0};
  query_result_type input_index(N);
  query_result_type results(N);
  LOG_DEBUG("Initialized functors.")
  thrust::sequence(input_index.begin(), input_index.end());
  thrust::transform(thrust::device, index, index + N, results.begin(),
                    find_functor);

  size_type const number_of_valid =
      thrust::remove_if(thrust::device,
                        thrust::make_zip_iterator(thrust::make_tuple(input_index.begin(), results.begin())),
                        thrust::make_zip_iterator(thrust::make_tuple(input_index.end(), results.end())),
                        invalid_functor) -
      thrust::make_zip_iterator(thrust::make_tuple(input_index.begin(), results.begin()));
  LOG_DEBUG("Number of valid", number_of_valid);
  input_index.resize(number_of_valid);
  results.resize(number_of_valid);

  return thrust::make_pair(input_index, results);
}

// Template instantiation
template class CoordinateMapGPU<int32_t>;
// Insert arg templates
using citer32 = coordinate_iterator<int32_t>;
template void
    CoordinateMapGPU<int32_t>::insert<thrust::counting_iterator<uint32_t>>(
        citer32, citer32, thrust::counting_iterator<uint32_t>,
        thrust::counting_iterator<uint32_t>);

} // namespace minkowski
