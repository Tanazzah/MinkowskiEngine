#include "hip/hip_runtime.h"
/* Copyright (c) 2020 NVIDIA CORPORATION.
 * Copyright (c) 2018-2020 Chris Choy (chrischoy@ai.stanford.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "coordinate_map_functors.cuh"
#include "coordinate_map_gpu.cuh"
#include "types.hpp"
#include "utils.hpp"

#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>

#include <torch/extension.h>

namespace minkowski {

using coordinate_type = int32_t;
using index_type = default_types::index_type;
using size_type = default_types::size_type;

size_type coordinate_map_batch_insert_test(const torch::Tensor &coordinates) {
  // Create TensorArgs. These record the names and positions of each tensor as a
  // parameter.
  torch::TensorArg arg_coordinates(coordinates, "coordinates", 0);

  torch::CheckedFrom c = "coordinate_test";
  torch::checkContiguous(c, arg_coordinates);
  // must match coordinate_type
  torch::checkScalarType(c, arg_coordinates, torch::kInt);
  torch::checkBackend(c, arg_coordinates.tensor, torch::Backend::CUDA);
  torch::checkDim(c, arg_coordinates, 2);

  auto const N = (index_type)coordinates.size(0);
  auto const D = (index_type)coordinates.size(1);
  coordinate_type const *d_ptr = coordinates.data_ptr<coordinate_type>();

  LOG_DEBUG("Initialize a GPU map.");
  CoordinateMapGPU<coordinate_type> map{N, D};

  auto input_coordinates = coordinate_range<coordinate_type>(N, D, d_ptr);
  thrust::counting_iterator<uint32_t> iter{0};

  LOG_DEBUG("Insert coordinates");
  map.insert(input_coordinates.begin(), // key begin
             input_coordinates.end(),   // key end
             iter,                      // value begin
             iter + N);                 // value end

  return map.size();
}

std::pair<std::vector<index_type>, std::vector<index_type>>
coordinate_map_batch_find_test(const torch::Tensor &coordinates,
                               const torch::Tensor &queries) {
  // Create TensorArgs. These record the names and positions of each tensor as a
  // parameter.
  torch::TensorArg arg_coordinates(coordinates, "coordinates", 0);
  torch::TensorArg arg_queries(queries, "queries", 1);

  torch::CheckedFrom c = "coordinate_test";
  torch::checkContiguous(c, arg_coordinates);
  torch::checkContiguous(c, arg_queries);

  // must match coordinate_type
  torch::checkScalarType(c, arg_coordinates, torch::kInt);
  torch::checkScalarType(c, arg_queries, torch::kInt);
  torch::checkBackend(c, arg_coordinates.tensor, torch::Backend::CUDA);
  torch::checkBackend(c, arg_queries.tensor, torch::Backend::CUDA);
  torch::checkDim(c, arg_coordinates, 2);
  torch::checkDim(c, arg_queries, 2);

  auto const N = (index_type)coordinates.size(0);
  auto const D = (index_type)coordinates.size(1);
  auto const NQ = (index_type)queries.size(0);
  auto const DQ = (index_type)queries.size(1);

  ASSERT(D == DQ, "Coordinates and queries must have the same size.");
  coordinate_type const *ptr = coordinates.data_ptr<coordinate_type>();
  coordinate_type const *query_ptr = queries.data_ptr<coordinate_type>();

  CoordinateMapGPU<coordinate_type> map{N, D};

  auto input_coordinates = coordinate_range<coordinate_type>(N, D, ptr);
  thrust::counting_iterator<uint32_t> iter{0};

  map.insert(input_coordinates.begin(), // key begin
             input_coordinates.end(),   // key end
             iter,                      // value begin
             iter + N);                 // value end

  LOG_DEBUG("Map size", map.size());
  auto query_coordinates = coordinate_range<coordinate_type>(NQ, D, query_ptr);

  LOG_DEBUG("Find coordinates.");
  auto const query_results =
      map.find(query_coordinates.begin(), query_coordinates.end());
  auto const &firsts(query_results.first);
  auto const &seconds(query_results.second);
  index_type NR = firsts.size();
  LOG_DEBUG(NR, "keys found.");

  std::vector<index_type> cpu_firsts(NR);
  std::vector<index_type> cpu_seconds(NR);

  thrust::copy(firsts.begin(), firsts.end(), cpu_firsts.begin());
  thrust::copy(seconds.begin(), seconds.end(), cpu_seconds.begin());
  return std::make_pair(cpu_firsts, cpu_seconds);
}

} // namespace minkowski

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("coordinate_map_batch_insert_test",
        &minkowski::coordinate_map_batch_insert_test,
        "Minkowski Engine coordinate map batch insert test");

  m.def("coordinate_map_batch_find_test",
        &minkowski::coordinate_map_batch_find_test,
        "Minkowski Engine coordinate map batch find test");
}
