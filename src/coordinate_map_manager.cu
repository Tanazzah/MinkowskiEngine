#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 NVIDIA CORPORATION.
 * Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "coordinate_map_gpu.cuh"
#include "coordinate_map_manager.cpp"
#include "coordinate_map_manager.hpp"

#include <ATen/cuda/HIPContext.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

namespace minkowski {

namespace detail {

template <typename SrcType, typename DstType>
__global__ void dtypeCopy(SrcType const *src, DstType *dst, size_t n) {
  CUDA_KERNEL_LOOP(index, n) { dst[index] = src[index]; }
}

template <typename coordinate_type,
          template <typename C> class TemplatedAllocator>
struct insert_and_map_functor<coordinate_type, TemplatedAllocator,
                              CoordinateMapGPU> {

  std::pair<at::Tensor, at::Tensor>
  operator()(coordinate_map_key_type &map_key, at::Tensor const &th_coordinate,
             CoordinateMapManager<coordinate_type, TemplatedAllocator,
                                  CoordinateMapGPU> &manager) {
    uint32_t const N = th_coordinate.size(0);
    uint32_t const coordinate_size = th_coordinate.size(1);
    coordinate_type *p_coordinate = th_coordinate.data_ptr<coordinate_type>();

    auto coordinate_map = CoordinateMapGPU<coordinate_type, TemplatedAllocator>(
        N, coordinate_size, DEFAULT_HASH_TABLE_OCCUPANCY, map_key.first);

    auto input_coordinate_range =
        coordinate_range<coordinate_type>(N, coordinate_size, p_coordinate);

    auto map_inverse_map = coordinate_map.insert_and_map(
        input_coordinate_range.begin(), input_coordinate_range.end());
    LOG_DEBUG("mapping size:", map_inverse_map.first.size());

    // insert moves map
    manager.insert(map_key, coordinate_map);

    auto const &mapping = map_inverse_map.first;
    auto const &inverse_mapping = map_inverse_map.second;

    // return tensors
    at::Tensor th_mapping =
        torch::empty({(int64_t)mapping.size()},
                     th_coordinate.options().requires_grad(false));
    at::Tensor th_inverse_mapping =
        torch::empty({(int64_t)inverse_mapping.size()},
                     th_coordinate.options().requires_grad(false));

    static_assert(sizeof(coordinate_type) == sizeof(default_types::index_type));
    CUDA_CHECK(hipMemcpy(th_mapping.data_ptr<coordinate_type>(),
                          thrust::raw_pointer_cast(mapping.data()),
                          mapping.size() * sizeof(default_types::index_type),
                          hipMemcpyDeviceToDevice));
    CUDA_CHECK(
        hipMemcpy(th_inverse_mapping.data_ptr<coordinate_type>(),
                   thrust::raw_pointer_cast(inverse_mapping.data()),
                   inverse_mapping.size() * sizeof(default_types::index_type),
                   hipMemcpyDeviceToDevice));

    return std::make_pair(std::move(th_mapping), std::move(th_inverse_mapping));
  }
};

} // namespace detail

template class CoordinateMapManager<int32_t, detail::default_allocator,
                                    CoordinateMapGPU>;
template class CoordinateMapManager<int32_t, detail::c10_allocator,
                                    CoordinateMapGPU>;

} // end namespace minkowski
